#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__
void vectorAdd(float* a, float* b, float* result, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result[i] = a[i] + b[i];
    }
}

int main() {
    const int size = 5;

    // Ініціалізація векторів на CPU
    float h_a[size] = { 1.0, 2.0, 3.0, 4.0, 5.0 };
    float h_b[size] = { 5.0, 4.0, 3.0, 2.0, 1.0 };
    float h_result[size];

    // Оголошення та алокація пам'яті на GPU
    float* d_a, * d_b, * d_result;
    hipMalloc((void**)&d_a, size * sizeof(float));
    hipMalloc((void**)&d_b, size * sizeof(float));
    hipMalloc((void**)&d_result, size * sizeof(float));

    // Копіювання даних з CPU на GPU
    hipMemcpy(d_a, h_a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size * sizeof(float), hipMemcpyHostToDevice);

    // Визначення конфігурації блоків та ниток
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;

    // Виклик ядра на GPU
    vectorAdd << <numBlocks, blockSize >> > (d_a, d_b, d_result, size);

    // Копіювання результату з GPU на CPU
    hipMemcpy(h_result, d_result, size * sizeof(float), hipMemcpyDeviceToHost);

    // Виведення результату
    for (int i = 0; i < size; ++i) {
        std::cout << h_result[i] << " ";
    }
    std::cout << std::endl;

    // Звільнення пам'яті на GPU
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return 0;
}
